#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <complex>
#include <cmath>
#include <fstream>
#include <sstream>
#include <string>
#include <omp.h>
#include <chrono>


const int SIZE = 70;

template <typename T>
__global__ void MatrixMulKernel(T* matA, T* matB, T* matOut, int size){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row<size&&col<size){
        T sumC= T(0);
        for (int k = 0; k < size; ++k)
            sumC += matA[row*SIZE+k] * matB[k*SIZE+Col];
        matOut[row*SIZE + col] = sumC;
    }
}


template <typename T>
void fillRandom(T mat[][SIZE], int size){
    for(int i = 0; i < size; i++)
        for(int j = 0; j < size; j++){
            T ran = (T)(rand()%20-10);
            // Niby niepotrzebne ale niech tu będzie
            while(ran == T(0)){
                ran = (T)(rand()%20-10);
            }
            mat[i][j] = ran;
        }
}

template <typename T>
void createIdentityMatrix(T mat[][SIZE], int size){
    for(int i = 0; i < size; i++)
        for(int j = 0; j < size; j++)
            mat[i][j] = (i == j) ? T(1) : T(0);
}

template <typename T>
void givensRotationMatrix(T mat[][SIZE], int i, int j, T x1, T x2){
    T s = T(1) / std::sqrt( x1 * x1 / (x2 * x2) + 1);
    T c = s * x1 / x2;
    mat[i][i]=c;
    mat[j][i]=T(-1) * s;
    mat[i][j]=s;
    mat[j][j]=c;
}

template <typename T>
void transpose(T mat[][SIZE], int size){
    T buff = T(0);
    for(int i=0;i<size;i++){
        for(int j=0;j<=i;j++){
            buff = mat[j][i];
            mat[j][i] = mat[i][j];
            mat[i][j] = buff;
        }
    }
}

template <typename T>
void copy(T mat1[][SIZE], T mat2[][SIZE], int size){
    for(int i=0;i<size;i++)
        for(int j=0;j<size;j++)
            mat2[i][j] = mat1[i][j];
}

template <typename T>
void multiply(T mat1[][SIZE],T mat2[][SIZE],T mat3[][SIZE], int size){
    for(int i =0;i<size;i++)
        for(int j=0;j<size;j++){
            T sum = T(0);
            for(int k = 0;k<size;k++)
                sum += mat1[i][k] * mat2[k][j];
            mat3[i][j] = sum;
        }
}

template <typename T>
bool compareToZero(T num){
    if(num < T(0.000001) && num > T(-0.000001))
        return true;
    return false;
}

// template <typename T>
// void QRDecomposeWithGivens(T mat1[][SIZE], T matQ[][SIZE], T matR[][SIZE], int size){
//     copy<T>(mat1,matR,size);

//     createIdentityMatrix<T>(matQ,size);

//     T tmp[SIZE][SIZE];
//     T rotMatrix[SIZE][SIZE];
//     for(int col = 0; col < size; col++){
//         for (int row = size - 1; row > col; row--){
//             createIdentityMatrix(rotMatrix,size);
//             givensRotationMatrix<T>(rotMatrix,row-1,row,matR[row-1][col],matR[row][col]);

//             printf("\nPierdolony progrman givens\n");
//             for(int i =0;i<size;i++){
//                 printf("\n");
//                 for(int j = 0; j<size;j++){
//                     printf("%.02f ",rotMatrix[i][j]);
//                 }
//             }

//             multiply(rotMatrix,matR,tmp,size);
//             copy(tmp,matR,size);

//             transpose(rotMatrix,size);
//             multiply(matQ,rotMatrix,tmp,size);
//             copy(tmp,matQ,size);
//         }
//     }
    
// }

template <typename T>
void QRDecomposeWithGivens(T mat1[][SIZE], T matQ[][SIZE], T matR[][SIZE], int size, dim3 gridSize,dim3 blockSize){
    copy<T>(mat1, matR, size);
    createIdentityMatrix<T>(matQ, size);
    T rotMatrix[SIZE][SIZE];
    T rotMatrixTr[SIZE][SIZE];
    T *d_rotMatrix,*d_rotMatrixTr, *d_matR, *d_matQ, *d_tmpR, *d_tmpQ;
    // Przydzielanie pamięci na GPU
    hipMalloc((void**)&d_rotMatrix, SIZE * SIZE * sizeof(T));
    hipMalloc((void**)&d_rotMatrixTr, SIZE * SIZE * sizeof(T));
    hipMalloc((void**)&d_matR, SIZE * SIZE * sizeof(T));
    hipMalloc((void**)&d_matQ, SIZE * SIZE * sizeof(T));
    hipMalloc((void**)&d_tmpR, SIZE * SIZE * sizeof(T));
    hipMalloc((void**)&d_tmpQ, SIZE * SIZE * sizeof(T));
    for(int col = 0; col < size; col++){
        for (int row = size - 1; row > col; row--){
            // Tworzenie Gi i Gi^T
            createIdentityMatrix(rotMatrix, size);
            createIdentityMatrix(rotMatrixTr, size);
            givensRotationMatrix<T>(rotMatrix, row-1, row, matR[row-1][col], matR[row][col]);
            givensRotationMatrix<T>(rotMatrixTr, row-1, row, matR[row-1][col], matR[row][col]);
            transpose(rotMatrixTr, size);
            //-- Iteracja R --
            // Kopiowanie danych do GPU
            hipMemcpy(d_rotMatrix, rotMatrix, SIZE * SIZE * sizeof(T), hipMemcpyHostToDevice);
            hipMemcpy(d_matR, matR, SIZE * SIZE * sizeof(T), hipMemcpyHostToDevice);
            // Mnożenie macierzy
            MatrixMulKernel<<<gridSize, blockSize>>>(d_rotMatrix, d_matR, d_tmpR, size);
            // Kopiowanie wyników z GPU na CPU
            hipMemcpy(matR, d_tmpR, SIZE * SIZE * sizeof(T), hipMemcpyDeviceToHost);
            //-- Iteracja Q --
            // Kopiowanie danych do GPU
            hipMemcpy(d_rotMatrixTr, rotMatrixTr, SIZE * SIZE * sizeof(T), hipMemcpyHostToDevice);
            hipMemcpy(d_matQ, matQ, SIZE * SIZE * sizeof(T), hipMemcpyHostToDevice);
            // // Mnożenie macierzy
            MatrixMulKernel<<<gridSize, blockSize>>>(d_matQ, d_rotMatrixTr, d_tmpQ, size);
            // Kopiowanie wyników z GPU na CPU
            hipMemcpy(matQ, d_tmpQ, SIZE * SIZE * sizeof(T), hipMemcpyDeviceToHost);
        }
    }
    // Zwolnienie pamięci GPU
    hipFree(d_rotMatrix);
    hipFree(d_matR);
    hipFree(d_tmpR);
    hipFree(d_tmpQ);
    hipFree(d_matQ);
}


template <typename T>
bool compareMatrices(T mat1[][SIZE],T mat2[][SIZE],int size){
    for(int i = 0; i<size; i++)
    for(int j = 0; j<size; j++)
        if(!compareToZero(mat1[i][j]-mat2[i][j]))
        return false;

    return true;
}

int main(){
    srand(100);
    const int n = SIZE;

    dim3 threadsPerBlock(n, n);
    dim3 blocksPerGrid(1, 1);
    if (n*n > 512){
        threadsPerBlock.x = 32;
        threadsPerBlock.y = 32;
        blocksPerGrid.x = ceil(double(n)/double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(n)/double(threadsPerBlock.y));
    }


    
    int nVal[20];
    int nInc = 3;
    int nStart = 9;
    for(int i = 0; i < 20; i ++){
        nVal[i] = nStart + nInc*i;
    }

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::ofstream file("resultsparcuda2.txt");
    if (!file.is_open()) {
        std::cerr << "Failed to open file." << std::endl;
        return 0;
    }
    for (size_t i = 0; i < 20; ++i) {
        file << nVal[i];
        if (i != 20 - 1)
            file << ";"; 
    }
    file << std::endl;

    
    for(int i = 0; i < 20; i++){
        double testMatrix[SIZE][SIZE];
        double testQ[SIZE][SIZE];
        double testR[SIZE][SIZE];
        fillRandom(testMatrix,SIZE);
    
        //printf("Checkpoint1\n");
        begin = std::chrono::steady_clock::now();
        QRDecomposeWithGivens(testMatrix,testQ,testR,nVal[i],blocksPerGrid,threadsPerBlock);
        end = std::chrono::steady_clock::now();

        int elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
        file << elapsed;
        if (i != 20 - 1)
            file << ";";

        std::cout << "Time difference = " << elapsed << "[ms]" << std::endl;
    }

    file.close();
    return 0;
}